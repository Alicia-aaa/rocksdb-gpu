#include "hip/hip_runtime.h"

#include <chrono>
#include <cstdio>
#include <functional>
#include <iostream>
#include <string>
#include <thread>
#include <vector>
extern "C" {
#include <pinpool.h>
#include <filemap.h>
}
#include "accelerator/cuda/block_decoder.h"
#include "accelerator/cuda/filter.h"
#include "rocksdb/slice.h"
#include "table/format.h"
#include "stdio.h"

#define KB 1024
#define MB 1024 * KB
#define GB 1024 * MB
#define MAX_DEPTH       24
#define SELECTION_SORT  16

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

namespace ruda {
void quickSortI(donardSlice* h_results, unsigned long long int* h_result_idx_arr, unsigned long long int left, unsigned long long int right);
void selection_sort(donardSlice *h_results, unsigned long long int* h_result_idx_arr, unsigned long long int left, unsigned long long int right); 
void quick_sort(donardSlice* h_results, unsigned long long int* h_result_idx_arr, unsigned long long int left, unsigned long long int right, int depth);
namespace kernel {
__global__
void rudaDonardFilterKernel(char **file_address, uint64_t size, uint64_t *block_index, uint64_t *g_block_index, uint64_t block_unit, uint64_t *handles,
 RudaSchema *schema, unsigned long long int *d_results_idx, donardSlice *d_results);

//__global__
//void rudaCopyKernel(unsigned long long int count, donardSlice *d_results, char* d_total_results, unsigned long long int *total_results_idx);

__global__
void rudaCopyKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *d_result_idx_arr, unsigned long long int *d_target_idx, char* d_total_results);

__global__
void makeIndex(unsigned long long int count, unsigned long long int *d_result_idx_arr);

__device__
char toHex(unsigned char v);

__device__
int memcmp_slice(const void *x, const void *y, size_t n); 

__device__
void selection_sort(donardSlice *d_results, unsigned long long int *d_result_idx_arr, unsigned long long int left, unsigned long long int right);

__global__
void cdp_simple_quicksort(donardSlice* d_results, unsigned long long int *d_result_idx_arr, unsigned long long int left, unsigned long long int right, int depth);

__global__
void testKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *total_results_idx);
}  // namespace kernel

struct DonardManager {

  // fileMap list
  struct filemap ** fmlist; 

  // Parameter

  unsigned long long int *num_entries_;

  int num_file_;
  int total_blocks_;
  int block_unit_;
  int num_thread_;
  int max_results_count_;   
  std::vector<uint64_t> gpu_blocks_;

  unsigned long long int results_size;
  unsigned long long int count;
 
  
  // MetaData
  char **file_address;
  uint64_t *block_index; // the number of blocks in each file
  uint64_t *g_block_index;
  uint64_t *d_handles;

  // Schema
  RudaSchema *d_schema; // device schema
  RudaSchema h_schema; // host schema

  // Result
  unsigned long long int* d_results_idx;
  donardSlice* d_results;
  donardSlice* h_results;

  unsigned long long int *total_results_idx;
  unsigned long long int *d_result_idx_arr;
  unsigned long long int *h_result_idx_arr;

  unsigned long long int *h_target_idx;
  unsigned long long int *d_target_idx;
  char * d_total_results;
 //char * h_total_results;

  DonardManager(int num_file, int total_blocks, int block_unit, int num_thread, int max_results_count) {
    //std::cout << "[DONARD MANAGER INITALIZE]" << std::endl;
    num_file_ = num_file;
    total_blocks_ = total_blocks;
    block_unit_ = block_unit;
    num_thread_ = num_thread;
    max_results_count_ = max_results_count;
    results_size = 0;
    count = 0;
  }

  void populate(std::vector<std::string> files, std::vector<uint64_t> num_blocks, std::vector<uint64_t> handles, const rocksdb::SlicewithSchema &schema) {
    //std::cout << "[DONARD POPULATE]" << std::endl;
    fmlist = (filemap **)malloc(sizeof(filemap *) * num_file_);
    for(uint i = 0; i < num_file_; i++) {
      fmlist[i] = filemap_open_cuda(files[i].c_str());
    }

    //std::cout << "[DONARD POPULATE1] " << num_file_ << " " << total_blocks_ << std::endl;
    //cudaCheckError(hipMalloc((void ***) &file_address, sizeof(char *) * num_file_));
    cudaCheckError(hipHostAlloc((void**)&file_address, sizeof(char*) * num_file_, hipHostMallocMapped));

    //std::cout << "[DONARD POPULATE 1-1]" << std::endl;
    for(uint i = 0; i < num_file_; i++) {
      file_address[i] = (char *)fmlist[i]->data;
    }
    
    int tmp_gpu_blocks = 0;
    for(uint i = 0; i < num_blocks.size(); i++) {
      int unit = 0;
      int remain = 0;
      if( i == 0 ) {
        unit = num_blocks[i] / block_unit_;
        remain = num_blocks[i] % block_unit_;
      } else {
        unit = (num_blocks[i] - num_blocks[i-1]) / block_unit_;
        remain = (num_blocks[i] - num_blocks[i-1]) % block_unit_;
      }
      tmp_gpu_blocks += unit;
      if(remain != 0) tmp_gpu_blocks += 1;
      gpu_blocks_.emplace_back(tmp_gpu_blocks);
    }

    //std::cout << "gpu block index size " << gpu_blocks_.size() << std::endl;

    //std::cout << "[DONARD POPULATE2] block_index " << sizeof(uint64_t) * num_blocks.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &block_index, sizeof(uint64_t) * num_blocks.size()));
    cudaCheckError(hipMemcpy(block_index, &num_blocks[0], sizeof(uint64_t) * num_blocks.size(), hipMemcpyHostToDevice));

    //std::cout << "[DONARD POPULATE2] g_block_index " << sizeof(uint64_t) * gpu_blocks_.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &g_block_index, sizeof(uint64_t) * gpu_blocks_.size()));
    cudaCheckError(hipMemcpy(g_block_index, &gpu_blocks_[0], sizeof(uint64_t) * gpu_blocks_.size(), hipMemcpyHostToDevice));

    //std::cout << "[DONARD POPULATE2] d_handles " << sizeof(uint64_t) * handles.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &d_handles, sizeof(uint64_t) * handles.size()));
    cudaCheckError(hipMemcpy(d_handles, &handles[0], sizeof(uint64_t) * handles.size(), hipMemcpyHostToDevice));
  
    //std::cout << "[DONARD POPULATE3]" << std::endl;
    // Deep copy for Schema
    rocksdb::SlicewithSchema* copy_schema = schema.clone();
    cudaCheckError(hipHostRegister(&h_schema, sizeof(rocksdb::SlicewithSchema), hipHostMallocMapped));
    cudaCheckError(hipMalloc((void **) &d_schema, sizeof(RudaSchema)));
    cudaCheckError(h_schema.populateToCuda(*copy_schema));
    cudaCheckError(hipMemcpy(d_schema, &h_schema, sizeof(RudaSchema), hipMemcpyHostToDevice));

    //std::cout << "[DONARD POPULATE4]" << std::endl;
    cudaCheckError(hipMalloc((void **) &d_results_idx, sizeof(unsigned long long int)));
    cudaCheckError(hipMemset(d_results_idx, 0, sizeof(unsigned long long int)));

   // std::cout << "[DONARD POPULATE5] d_results " << sizeof(donardSlice) * max_results_count_ << std::endl;
    cudaCheckError(hipMalloc((void **) &d_results, sizeof(donardSlice) * max_results_count_));

    cudaCheckError(hipMalloc((void **) &total_results_idx, sizeof(unsigned long long int)));
    cudaCheckError(hipMemset(total_results_idx, 0, sizeof(unsigned long long int)));
  
  }

  void executeKernel(double *pushdown_evaluate) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //std::cout << "Filterig" << std::endl;
    hipEventRecord(start);
    //std::cout << "[DONARD KERNEL EXECUTE] : " << gpu_blocks_.back() << std::endl;
    kernel::rudaDonardFilterKernel<<< gpu_blocks_.back(), num_thread_ >>> (file_address, num_file_, block_index, g_block_index, block_unit_, d_handles,
                             d_schema, d_results_idx, d_results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    *pushdown_evaluate = (double) milliseconds; 

    hipDeviceSynchronize();

    num_entries_ = (unsigned long long int *)malloc(sizeof(unsigned long long int));
    cudaCheckError(hipMemcpy(num_entries_, d_results_idx, sizeof(unsigned long long int), hipMemcpyDeviceToHost));

    count = *num_entries_;
    uint32_t blockGrid = count / num_thread_ ;
    uint32_t remain = count % num_thread_ ;   
    if (remain != 0) blockGrid += 1;

   /**************************SORT IMPLEMENTATION START****************************/
    //cudaCheckError(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
  
    //unsigned long long int left = 0;
    //unsigned long long int right = *num_entries_ - 1;
    //std::cout << "Make INDEX " << std::endl;
    //cudaCheckError(hipMalloc((void **) &d_result_idx_arr, sizeof(unsigned long long int) * (right + 1)));
    //kernel::makeIndex<<< blockGrid, num_thread_ >>> (count, d_result_idx_arr);

    //std::cout << "Sorting on the GPU : " << right << std::endl;
    //kernel::cdp_simple_quicksort<<<1, 1>>>(d_results, d_result_idx_arr, left, right, 0);
    //cudaCheckError(hipDeviceSynchronize());

   /**************************SORT IMPLEMENTATION END****************************/

    unsigned long long int left = 0;
    unsigned long long int right = count;
    h_results = (donardSlice *)malloc(sizeof(donardSlice) * count);
    cudaCheckError(hipMemcpy(h_results, d_results, sizeof(donardSlice) * count, hipMemcpyDeviceToHost));

    h_result_idx_arr = (unsigned long long int *)malloc(sizeof(unsigned long long int) * count);
    for(int i = 0; i < count; i++) h_result_idx_arr[i] = i;
    // std::cout << "Sorting on the CPU Quick : " << count << std::endl;
    // selection_sort(h_results, h_result_idx_arr, left, right-1);
    // quick_sort(h_results, h_result_idx_arr, left, right-1, 0);
    quickSortI(h_results, h_result_idx_arr, left, right - 1);
    h_target_idx = (unsigned long long int *)malloc(sizeof(unsigned long long int) * count);

    for(uint i = 0; i < count ; i++) {
      if (i == 0)
        h_target_idx[i] = 0;
      else 
        h_target_idx[i] = h_target_idx[i-1] + h_results[h_result_idx_arr[i-1]].key_size + h_results[h_result_idx_arr[i-1]].d_size + 4;
      results_size += h_results[i].key_size + h_results[i].d_size + 4;
    }

    //std::cout << "Copying Device to Host" << std::endl;

    cudaCheckError(hipMalloc((void **) &d_result_idx_arr, sizeof(unsigned long long int) * count));
    cudaCheckError(hipMemcpy(d_result_idx_arr, h_result_idx_arr, sizeof(unsigned long long int) * count, hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void **) &d_target_idx, sizeof(unsigned long long int) * count));
    cudaCheckError(hipMemcpy(d_target_idx, h_target_idx, sizeof(unsigned long long int) * count, hipMemcpyHostToDevice));

    //std::cout << " results_size : " << results_size << std::endl;
    cudaCheckError(hipMalloc((void **) &d_total_results, sizeof(char) * results_size));


    //std::cout << " blockGrid : " << blockGrid << " count : " << count << std::endl;
    //kernel::rudaCopyKernel<<< blockGrid , num_thread_ >>> (count, d_results, d_total_results, total_results_idx); 
    kernel::rudaCopyKernel<<< blockGrid , num_thread_ >>> (count, d_results, d_result_idx_arr, d_target_idx, d_total_results);
    hipDeviceSynchronize();
  }

  void translatePairsToSlices(std::vector<rocksdb::PinnableSlice> &keys, std::vector<rocksdb::PinnableSlice> &results, char **data_buf, uint64_t *num_entries) {

    //std::cout << "[DONARD TRANSLATE TO SLICES 0]" << std::endl;
    //h_total_results = (char *)malloc(sizeof(char) * results_size);
    //cudaCheckError(hipMemcpy(h_total_results, d_total_results, sizeof(char) * results_size, hipMemcpyDeviceToHost));

    *num_entries = count;
    *data_buf = (char *)malloc(sizeof(char) * results_size);
    char *target_ptr = *data_buf;
    cudaCheckError(hipMemcpy(target_ptr, d_total_results, sizeof(char) * results_size, hipMemcpyDeviceToHost));
  
    /*
    //std::cout << "[DONARD TRANSLATE TO SLICES 1]" << std::endl;  
    //std::cout << "[DONARD TRANSLATE TO SLICES 2] " << count << std::endl;

    char *initialPtr = h_total_results;
    for (size_t i = 0; i < count; i++) {
      size_t key_size = *((unsigned short *)initialPtr);
      initialPtr += 2;
      size_t value_size = *((unsigned short *)initialPtr);
      initialPtr += 2;

      keys.emplace_back(std::move(rocksdb::PinnableSlice(initialPtr, key_size)));
      initialPtr += key_size;

      results.emplace_back(std::move(rocksdb::PinnableSlice(initialPtr, value_size)));
      initialPtr += value_size;
    }
   */
  }

  void clear() {
    for(uint i = 0; i < num_file_; i++) {
      filemap_free(fmlist[i]);
    } 
    cudaCheckError(hipHostFree(file_address)); 
    cudaCheckError(hipFree(block_index));
    cudaCheckError(hipFree(g_block_index));
    cudaCheckError(hipFree(d_handles));

    cudaCheckError(h_schema.clear());
    cudaCheckError(hipFree(d_schema));

    cudaCheckError(hipFree(d_results_idx));
    cudaCheckError(hipFree(d_results));
    cudaCheckError(hipFree(total_results_idx));
    cudaCheckError(hipFree(d_result_idx_arr));
    cudaCheckError(hipFree(d_total_results));
    cudaCheckError(hipFree(d_target_idx));

    free(h_target_idx);
    free(h_results);
    free(h_result_idx_arr);
   // free(h_total_results);
  }
};

__global__
void kernel::rudaDonardFilterKernel(char **file_address, uint64_t size, uint64_t *block_index, uint64_t *g_block_index, uint64_t g_block_unit, uint64_t * d_handles,
 RudaSchema *schema, unsigned long long int *results_idx, donardSlice *d_results) {  
  
  // blockDim.x * blockIdx.x + threadIdx.x;
  // blockDim = number of Thread in block

  // Find file location 
  unsigned int idx = getFileIdx(blockIdx.x, size, g_block_index);

  int gBlockOffset = (idx == 0) ? blockIdx.x : blockIdx.x - g_block_index[idx-1];
  int accumulatedBlocks = (idx == 0) ? g_block_unit * gBlockOffset : block_index[idx-1] + g_block_unit * gBlockOffset;

  int gBlockRemain = 0;
  if (blockIdx.x == g_block_index[idx] - 1) {
    gBlockRemain = (idx == 0) ? block_index[idx] % g_block_unit : (block_index[idx] - block_index[idx -1]) % g_block_unit;
  }
  if (gBlockRemain != 0) g_block_unit = gBlockRemain; 

  char *filePtr = file_address[idx];  
  const char *startPtr = (gBlockOffset == 0) ? filePtr : filePtr + d_handles[accumulatedBlocks -1];

  uint32_t blockSize = 0;
  int kDataBlockIndexTypeBitShift = 31;
  uint32_t kNumRestartsMask = (1u << kDataBlockIndexTypeBitShift) - 1u;
  uint32_t kBlockTrailerSize = 5;

  uint32_t threadsPerBlock = blockDim.x / g_block_unit;
  uint32_t threadRemain = blockDim.x % g_block_unit;
  uint32_t threadIdInBlock = threadIdx.x / g_block_unit;
  uint32_t blockLocation = threadIdx.x % g_block_unit; 

  if (blockLocation < threadRemain) threadsPerBlock += 1;

  if (gBlockOffset == 0) {
   if(blockLocation == 0) {
    blockSize = d_handles[accumulatedBlocks] - kBlockTrailerSize;
   } else {
    blockSize = d_handles[accumulatedBlocks + blockLocation] - d_handles[accumulatedBlocks + blockLocation - 1] - kBlockTrailerSize;
    startPtr += d_handles[accumulatedBlocks + blockLocation - 1];
   }
  } else { 
    blockSize = d_handles[accumulatedBlocks + blockLocation] - d_handles[accumulatedBlocks + blockLocation - 1] - kBlockTrailerSize; 
    startPtr += d_handles[accumulatedBlocks + blockLocation - 1] - d_handles[accumulatedBlocks - 1];
  }

  uint32_t numRestarts = DecodeFixed32(startPtr + blockSize - sizeof(uint32_t));
  numRestarts = numRestarts & kNumRestartsMask;
  uint32_t restartOffset = static_cast<uint32_t>(blockSize) - (1 + numRestarts) * sizeof(uint32_t);

  if (numRestarts < threadIdInBlock + 1) return;

  uint32_t numTask = numRestarts / threadsPerBlock;
  uint32_t remainNumTask = numRestarts % threadsPerBlock;

  bool lastThread = false;
  if (numTask == 0 && threadIdInBlock == numRestarts - 1) lastThread = true;
  if (numTask != 0 && threadIdInBlock == threadsPerBlock - 1) lastThread = true; 

  if (threadIdInBlock < remainNumTask) numTask += 1;

  if(numTask == 0) return;

  uint32_t startLocation = restartOffset;

  startLocation += (threadIdInBlock >= remainNumTask) ? (remainNumTask + (numTask * threadIdInBlock)) * sizeof(uint32_t) : (numTask * threadIdInBlock * sizeof(uint32_t));

  if (!lastThread) restartOffset = 0;
  DecodeNFilterOnSchemaDonard(startPtr, restartOffset, startLocation, numTask, schema, results_idx, d_results); 

}

/*
__global__
void kernel::rudaCopyKernel(unsigned long long int count, donardSlice *d_results, char* total_results, unsigned long long int *total_results_idx) {

  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx >= count) {
    return;
  }

  //printf("blockidx : %d, threadidx : %d\n", blockIdx.x, threadIdx.x);
  size_t key_size = d_results[idx].key_size;
  size_t value_size = d_results[idx].d_size;
  unsigned long long int kvPairSize = key_size + value_size;

  unsigned long long int resultOffset = atomicAdd(total_results_idx, kvPairSize + 4);

  char* targetIdx = total_results + resultOffset;

  char *k_size = (char *)&key_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = k_size[i];
  }

  targetIdx += 2;

  char *v_size = (char *)&value_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = v_size[i];
  }

  targetIdx += 2;

  for(uint i = 0; i < key_size; i++) {
    targetIdx[i] = d_results[idx].key[i];
  }

  targetIdx += key_size;

  for(uint i = 0; i < value_size; i++) {
    targetIdx[i] = d_results[idx].d_data[i];
  }  
} */

__global__
void kernel::rudaCopyKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *d_result_idx_arr,
                            unsigned long long int *d_target_idx, char* total_results) {

  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx >= count) {
    return;
  }

  //printf("blockidx : %d, threadidx : %d\n", blockIdx.x, threadIdx.x);
  size_t key_size = d_results[d_result_idx_arr[idx]].key_size;
  size_t value_size = d_results[d_result_idx_arr[idx]].d_size;

  char* targetIdx = total_results + d_target_idx[idx];

  char *k_size = (char *)&key_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = k_size[i];
  }

  targetIdx += 2;

  char *v_size = (char *)&value_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = v_size[i];
  }

  targetIdx += 2;

  for(uint i = 0; i < key_size; i++) {
    targetIdx[i] = d_results[d_result_idx_arr[idx]].key[i];
  }

  targetIdx += key_size;

  for(uint i = 0; i < value_size; i++) {
    targetIdx[i] = d_results[d_result_idx_arr[idx]].d_data[i];
  }  
}

__global__
void kernel::makeIndex(unsigned long long int count, unsigned long long int* d_result_idx_arr) {
  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx >= count) {
    return;
  }

  d_result_idx_arr[idx] = idx;
}


__device__
char kernel::toHex(unsigned char v) {
  if (v <= 9) {
    return '0' + v;
  }
  return 'A' + v - 10;
}

__device__
int kernel::memcmp_slice(const void* s1, const void* s2, size_t n) {
  const unsigned char *p1 = (const unsigned char *) s1;
  const unsigned char *p2 = (const unsigned char *) s2;
  p1 += 4;
  p2 += 4;
  n -= 4;
  while(n--)
    if( *p1 != *p2 )
      return *p1 - *p2;
    else
      p1++,p2++;
    return 0;
}
/*
__device__ 
void kernel::selection_sort(donardSlice *d_results, unsigned long long int* d_result_idx_arr,
                            unsigned long long int left, unsigned long long int right) {
  for( int i = left ; i <= right ; ++i ) {
    int min_val = d_results[d_result_idx_arr[i]].k_int;
    int min_idx = i;

    for( int j = i+1 ; j <= right ; ++j ) {
      int val_j = d_results[d_result_idx_arr[j]].k_int;
      if (val_j < min_val) {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if( i != min_idx ) {
      unsigned long long int temp = d_result_idx_arr[i]; 
      d_result_idx_arr[i] = d_result_idx_arr[min_idx];
      d_result_idx_arr[min_idx] = temp;
    }
  }
}

__global__
void kernel::cdp_simple_quicksort(donardSlice* d_results, unsigned long long int* d_result_idx_arr,
             unsigned long long int left, unsigned long long int right, int depth) {
  if( depth >= MAX_DEPTH || right - left <= SELECTION_SORT ) {
    selection_sort(d_results, d_result_idx_arr, left, right);
    return;
  }

  unsigned long long int *lptr = d_result_idx_arr + left;
  unsigned long long int *rptr = d_result_idx_arr + right;
  unsigned long long int pivot = d_result_idx_arr[(left + right)/2];

  while (lptr <= rptr) {
    int lval = d_results[(*lptr)].k_int;
    int rval = d_results[(*rptr)].k_int;
    int pval = d_results[pivot].k_int;

    // Move the left pointer as long as the pointed element is smaller than the pivot.
    while (lval < pval) {
      lptr++;
      lval = d_results[(*lptr)].k_int;
    }

    // Move the right pointer as long as the pointed element is larger than the pivot.
    while (rval > pval) {
      rptr--;
      rval = d_results[(*rptr)].k_int;
    }

    // If the swap points are valid, do the swap!
    if (lptr <= rptr) {
      unsigned long long int temp = *lptr;
      *lptr = *rptr;
      *rptr = temp;
      lptr++;
      rptr--;
    }
  }

  unsigned long long int nright = rptr - d_result_idx_arr;
  unsigned long long int nleft  = lptr - d_result_idx_arr;

  if (left < (rptr - d_result_idx_arr)) {
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    cdp_simple_quicksort<<< 1, 1, 0, s >>>(d_results, d_result_idx_arr, left, nright, depth+1);
    hipStreamDestroy(s);
  }

  if ((lptr - d_result_idx_arr) < right) {
    hipStream_t s1;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(d_results, d_result_idx_arr, nleft, right, depth+1);
    hipStreamDestroy(s1);
  }
}
*/
__global__
void kernel::testKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *total_results_idx) {
  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;
  printf("idx : %d\n", idx);
}

int compare(unsigned int a[], unsigned int b[], int pk_num) {
  for(int i = 0; i < pk_num; i++) {
    if (a[i] < b[i]) return -1;
    if (a[i] > b[i]) return 1;
  } 
  return 0;
}

void selection_sort(donardSlice *h_results, unsigned long long int* h_result_idx_arr,
                            unsigned long long int left, unsigned long long int right) {
  for( int i = left ; i <= right ; ++i ) {
    char* min_val = h_results[h_result_idx_arr[i]].key;
    int min_idx = i;

    for( int j = i+1 ; j <= right ; ++j ) {
      char* val_j = h_results[h_result_idx_arr[j]].key;
      if (val_j < min_val) {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if( i != min_idx ) {
      unsigned long long int temp = h_result_idx_arr[i]; 
      h_result_idx_arr[i] = h_result_idx_arr[min_idx];
      h_result_idx_arr[min_idx] = temp;
    }
  }
}

void quick_sort(donardSlice* h_results, unsigned long long int* h_result_idx_arr, 
                unsigned long long int left, unsigned long long int right, int depth) {
 
  if(depth % 1000 == 0) std::cout << "depth : " << depth << std::endl;
  int key_size = h_results[0].key_size;

  unsigned long long int* lptr = h_result_idx_arr + left;
  unsigned long long int* rptr = h_result_idx_arr + right;
  unsigned long long int pivot = h_result_idx_arr[(left + right)/ 2];

  char* lval;
  char* rval;
  char* pval;

  while(lptr <= rptr) {
    lval = h_results[(*lptr)].key;
    rval = h_results[(*rptr)].key;
    pval = h_results[pivot].key;

    while(memcmp(lval, pval, key_size) < 0 && ((lptr - h_result_idx_arr) < right)) {
      lptr++;
      lval = h_results[(*lptr)].key;
    }

    while(memcmp(rval, pval, key_size) > 0 && ((rptr - h_result_idx_arr) > left)) {
      rptr--;
      rval = h_results[(*rptr)].key;
    }

    if(lptr <= rptr) {
      unsigned long long int temp = *lptr;
      *lptr = *rptr;
      *rptr = temp;
      lptr++;
      rptr--;
    }
  }

  unsigned long long int nright = rptr - h_result_idx_arr;
  unsigned long long int nleft  = lptr - h_result_idx_arr;

  if(left < nright) {
    quick_sort(h_results, h_result_idx_arr, left, nright, depth+1);
  }

  if(nleft < right) {
    quick_sort(h_results, h_result_idx_arr, nleft, right, depth+1);
  }
}

typedef unsigned long long item;

typedef struct stack
{
  uint32_t size;
  uint32_t top;
  item *entries;
} stack;

# define MIN_STACK 128
# define INVALID   0xDeadD00d
# define SWAP(x, y)     { unsigned long long int t = x; x = y; y = t; }

stack *newStack() {
  stack *s = (stack *) calloc(MIN_STACK, sizeof(stack));
  if (s) {
    s->size = MIN_STACK;
    s->top = 0;
    s->entries = (item *) calloc(MIN_STACK, sizeof(item));
    if (s->entries) {
      return s;
    }
  }
  return (stack *)0;
}

void delStack(stack *s) {
  if (s->entries) {
    free(s->entries);
  }
  if (s) {
    free(s);
  }
  return;
}

item pop(stack *s) {
  if (s && s->top > 0) {
    s->top -= 1;
    return s->entries[s->top];
  }
  return INVALID;
}

void push(stack *s, item i) {
  if (s) {
    if (s->top == s->size) {
      s->size *= 2;
      s->entries = (item *) realloc(s->entries, s->size * sizeof(item));
    }
    s->entries[s->top] = i;
    s->top += 1;
  }
  return;
}

bool empty(stack *s) { return s->top == 0; }

unsigned long long int partition(donardSlice* h_results, unsigned long long int* h_result_idx_arr, 
              unsigned long long int left, unsigned long long int right, int key_size) {
  unsigned long long int pivotValue = h_result_idx_arr[(left + right) / 2];

  unsigned long long int i = left - 1;
  unsigned long long int j = right + 1;
  do {
    do {
      i += 1;
    } while (memcmp(h_results[h_result_idx_arr[i]].key, h_results[pivotValue].key, key_size) < 0);
    do {
      j -= 1;
    } while (memcmp(h_results[h_result_idx_arr[j]].key, h_results[pivotValue].key, key_size) > 0);
    if (i < j) {
      SWAP(h_result_idx_arr[i], h_result_idx_arr[j]);
    }
  } while (i < j);
  return j;
}

void quickSortI(donardSlice* h_results, unsigned long long int* h_result_idx_arr, 
                unsigned long long int left, unsigned long long int right) {
  stack *s = newStack();

  int key_size = h_results[0].key_size;
  push(s, left);
  push(s, right);
  while (!empty(s)) {
    unsigned long long int high = pop(s);
    unsigned long long int low = pop(s);
    unsigned long long int p = partition(h_results, h_result_idx_arr, low, high, key_size);
    if (p + 1 < high) {
      push(s, p + 1);
      push(s, high);
    }
    if (low < p) {
      push(s, low);
      push(s, p);
    }
  }
  delStack(s);
  return;
}

int donardFilter( std::vector<std::string> files, std::vector<uint64_t> num_blocks, std::vector<uint64_t> handles, const rocksdb::SlicewithSchema &schema,
                  uint64_t max_results_count,
                  std::vector<rocksdb::PinnableSlice> &keys,
                  std::vector<rocksdb::PinnableSlice> &results, char **data_buf, uint64_t *num_entries, double* pushdown_evaluate) {

  // std::cout << "[GPU][donardFilter] START" << std::endl;

  void *warming_up;
  cudaCheckError(hipMalloc(&warming_up, 0));
  cudaCheckError(hipFree(warming_up));

  DonardManager donard_mgr(
      files.size(),
      num_blocks.back(),
      30,
      128 /* kBlockSize */,
      max_results_count);

  donard_mgr.populate(files, num_blocks, handles, schema);

  donard_mgr.executeKernel(pushdown_evaluate);

  donard_mgr.translatePairsToSlices(keys, results, data_buf, num_entries);
  donard_mgr.clear();

//  std::cout << "This is end " << std::endl;
  hipDeviceSynchronize();
  hipDeviceReset();
  return accelerator::ACC_OK;
}

}  // namespace ruda
