#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
#include <string>
#include <vector>

#include "accelerator/cuda/block_decoder.h"
#include "accelerator/cuda/filter.h"
#include "rocksdb/slice.h"
#include "table/format.h"

#define KB 1024
#define MB 1024 * KB
#define GB 1024 * MB

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

namespace ruda {
namespace kernel {
// Kernels
__global__
void rudaStreamIntBlockFilterKernel(// Parameters (ReadOnly)
                                    size_t offset, size_t kSize,
                                    size_t dataSize, size_t maxCacheSize,
                                    char *data, uint64_t *seek_indices,
                                    accelerator::FilterContext *ctx,
                                    uint64_t *block_seek_start_indices,
                                    // Variables
                                    unsigned long long int *results_idx,
                                    // Results
                                    RudaKVPair *results);
}  // namespace kernel

struct RudaBlockStreamContext {
  hipStream_t stream;

  // Cuda Kernel Parameters
  const size_t kSize = 0;             // Total seek indices count
  const int kBlockSize = 0;
  const int kGridSize = 0;
  const size_t kMaxResultsCount = 0;  // Total count of filtered Key-Value pairs
  size_t kApproxResultsCount = 0;

  // Cuda Kernel Parameters - stream
  const int kStreamCount = 0;
  const int kStreamSize = 0;
  const int kGridSizePerStream = 0;

  // Cuda Results - Device
  RudaKVPair *d_results;                  // Filtered KV pairs
  unsigned long long int *d_results_idx;  // Atomic increment counter index

  // Cuda Results - Host
  // Total results count copied from 'd_results_idx' after kernel call...
  RudaKVPair *h_results;
  unsigned long long int h_results_count;

  // Max cached datablocks size on same gpu block (For using SharedMemory)
  size_t kMaxCacheSize = 0;

  // Allocated offsets on gpu
  size_t seek_start_offset, seek_size, datablocks_start_offset, datablocks_size;

  uint64_t *d_gpu_block_seek_starts;
  uint64_t *gpu_block_seek_starts;

  // Log
  size_t total_gpu_used_memory = 0;

  RudaBlockStreamContext(const size_t total_size, const int block_size,
                         const int grid_size, const size_t max_results_count,
                         const int stream_count, const int stream_size,
                         const int grid_size_per_stream)
      : kSize(total_size), kBlockSize(block_size), kGridSize(grid_size),
        kMaxResultsCount(max_results_count), kStreamCount(stream_count),
        kStreamSize(stream_size), kGridSizePerStream(grid_size_per_stream) {
    cudaCheckError(hipHostMalloc(
        (void **) &gpu_block_seek_starts,
        sizeof(uint64_t) * kGridSizePerStream));
    cudaCheckError(hipMalloc(
        (void **) &d_results_idx, sizeof(unsigned long long int)));
    total_gpu_used_memory += sizeof(unsigned long long int);
    kApproxResultsCount = kMaxResultsCount / (kStreamCount - 1);
    cudaCheckError(hipMalloc(
        (void **) &d_results, sizeof(RudaKVPair) * kApproxResultsCount));
    total_gpu_used_memory += sizeof(RudaKVPair) * kApproxResultsCount;
    cudaCheckError(hipHostMalloc(
        (void **) &h_results, sizeof(RudaKVPair) * kApproxResultsCount));
  }

  void cudaMallocGpuBlockSeekStarts() {
    cudaCheckError(hipMalloc(
        (void **) &d_gpu_block_seek_starts,
        sizeof(uint64_t) * kGridSizePerStream));
    total_gpu_used_memory += sizeof(uint64_t) * kGridSizePerStream;
  }

  size_t calculateGpuBlockSeekStarts(const std::vector<char> &datablocks,
                                     const std::vector<uint64_t> &seek_indices,
                                     size_t start, size_t size) {
    for (size_t i = 0; i < kGridSizePerStream; ++i) {
      size_t thread_idx = start + i * kBlockSize;
      if (thread_idx >= kSize) {
        gpu_block_seek_starts[i] = 0;
      } else {
        gpu_block_seek_starts[i] = seek_indices[thread_idx];
      }
    }

    size_t max_cache_size = 0;
    for (size_t i = 0; i < kGridSizePerStream; ++i) {
      size_t thread_idx = start + i * kBlockSize;
      if (thread_idx >= kSize) {
        break;
      }

      size_t cache_size;
      if (start + size == kSize) {
        // Last Stream case
        size_t next_block_thread_idx = start + (i + 1) * kBlockSize;
        if (next_block_thread_idx >= kSize) {
          cache_size = datablocks.size() - gpu_block_seek_starts[i];
        } else {
          cache_size = gpu_block_seek_starts[i+1] - gpu_block_seek_starts[i];
        }
      } else {
        // Non-last Stream case
        if (i == kGridSizePerStream - 1) {
          cache_size = seek_indices[start + size] - gpu_block_seek_starts[i];
        } else {
          cache_size = gpu_block_seek_starts[i+1] - gpu_block_seek_starts[i];
        }
      }

      if (cache_size > max_cache_size) {
        max_cache_size = cache_size;
      }
    }
    return sizeof(char) * max_cache_size;
  }

  void initParams(const std::vector<char> &datablocks,
                  const std::vector<uint64_t> &seek_indices,
                  size_t start, size_t size, size_t start_datablocks,
                  size_t size_datablocks) {
    seek_start_offset = start;
    seek_size = size;
    datablocks_start_offset = start_datablocks;
    datablocks_size = size_datablocks;
    kMaxCacheSize = calculateGpuBlockSeekStarts(
        datablocks, seek_indices, start, size);
  }

  void populateToCuda(const std::vector<char> &datablocks,
                      const std::vector<uint64_t> &seek_indices,
                      char *d_datablocks, uint64_t *d_seek_indices) {
    populateToCuda_d_results_idx();
    populateToCuda_d_datablocks(datablocks, d_datablocks);
    populateToCuda_d_seek_indices(seek_indices, d_seek_indices);
    populateToCuda_d_gpu_block_seek_starts();
  }

  void populateToCuda_d_results_idx() {
    cudaCheckError(hipMemsetAsync(
        d_results_idx, 0, sizeof(unsigned long long int), stream));
  }

  void populateToCuda_d_datablocks(const std::vector<char> &datablocks,
                                   char *d_datablocks) {
    cudaCheckError(hipMemcpyAsync(
        &d_datablocks[datablocks_start_offset],
        &datablocks[datablocks_start_offset],
        sizeof(char) * datablocks_size, hipMemcpyHostToDevice,
        stream));
  }

  void populateToCuda_d_seek_indices(const std::vector<uint64_t> &seek_indices,
                                     uint64_t *d_seek_indices) {
    cudaCheckError(hipMemcpyAsync(
        &d_seek_indices[seek_start_offset], &seek_indices[seek_start_offset],
        sizeof(uint64_t) * seek_size, hipMemcpyHostToDevice,
        stream));
  }

  void populateToCuda_d_gpu_block_seek_starts() {
    cudaCheckError(hipMemcpyAsync(
        d_gpu_block_seek_starts, gpu_block_seek_starts,
        sizeof(uint64_t) * kGridSizePerStream, hipMemcpyHostToDevice,
        stream));
  }

  void executeKernel(// Kernel Parameter
                     size_t kTotalDataSize,
                     // Sources
                     char *d_datablocks, uint64_t *d_seek_indices,
                     accelerator::FilterContext *d_cond_ctx) {
    kernel::rudaStreamIntBlockFilterKernel<<<kGridSizePerStream,
                                             kBlockSize,
                                             kMaxCacheSize,
                                             stream>>>(
      seek_start_offset, kSize, kTotalDataSize, kMaxCacheSize,
      d_datablocks, d_seek_indices, d_cond_ctx, d_gpu_block_seek_starts,
      d_results_idx, d_results
    );
  }

  void copyFromCuda() {
    cudaCheckError(hipMemcpyAsync(
        &h_results_count, d_results_idx, sizeof(unsigned long long int),
        hipMemcpyDeviceToHost, stream));
    cudaCheckError(hipMemcpyAsync(
        h_results, d_results, sizeof(RudaKVPair) * h_results_count,
        hipMemcpyDeviceToHost, stream));
  }

  void freeCudaObjects() {
    cudaCheckError( hipFree(d_gpu_block_seek_starts) );
    cudaCheckError( hipFree(d_results_idx) );
    cudaCheckError( hipFree(d_results) );
  }

  void initializeStream() {
    cudaCheckError( hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) );
  }

  void destroyStream() {
    cudaCheckError( hipStreamDestroy(stream) );
  }

  void clear() {
    freeCudaObjects();
    destroyStream();
    cudaCheckError( hipHostFree(gpu_block_seek_starts) );
    cudaCheckError( hipHostFree(h_results) );
  }
};

struct RudaBlockStreamManager {
  // Cuda Kernel Parameters
  // IMPORTANT: Kernel Parameters never be changed except in constructor.
  size_t kSize = 0;             // Total seek indices count
  int kBlockSize = 0;
  int kGridSize = 0;
  size_t kMaxResultsCount = 0;  // Total count of filtered Key-Value pairs

  // Cuda Kernel Parameters - stream
  int kStreamCount = 0;
  int kApproxStreamSize = 0;
  int kApproxGridSizePerStream = 0;

  // Streams
  std::vector<RudaBlockStreamContext> stream_ctxs;

  // Parameters
  char *d_datablocks;
  uint64_t *d_seek_indices;
  accelerator::FilterContext *d_cond_ctx;

  // Log
  size_t total_gpu_used_memory = 0;

  RudaBlockStreamManager(const size_t total_size, const int block_size,
                         const size_t stream_count,
                         const size_t max_results_count) {
    kSize = total_size;
    kBlockSize = block_size;
    kStreamCount = stream_count;
    kMaxResultsCount = max_results_count;
    size_t threads_per_stream = ceil((float) total_size / (float) stream_count);
    while (threads_per_stream <= kBlockSize && kBlockSize != 4) {
      kBlockSize = kBlockSize >> 1;
    }
    kGridSize = ceil((float) total_size / (float) kBlockSize);

    // Stream grid pre-process
    // ex) kSize = 672, kGridSize = 11, kBlockSize = 64, kStreamCount = 4
    // --> kApproxGridSizePerStream = 2, kApproxStreamSize = 128
    //
    // Stream1   { kGridSizePerStream = 2, kStreamSize = 128 }
    // Stream2   { kGridSizePerStream = 3, kStreamSize = 192 }
    // Stream3   { kGridSizePerStream = 3, kStreamSize = 192 }
    // Stream4   { kGridSizePerStream = 3, kStreamSize = 192 (actually 160) }
    // -----Results-----
    // <Stream1> <Stream2>    <Stream3>     <Stream4>
    // [64][64]  [64][64][64] [64][64][64]  [64][64][32]
    kApproxGridSizePerStream = kGridSize / kStreamCount;
    size_t additional_grid_count = kGridSize % kStreamCount;
    kApproxStreamSize = kApproxGridSizePerStream * kBlockSize;
    for (size_t i = 0; i < kStreamCount; ++i) {
      size_t grid_size_per_stream, stream_size;
      if (i >= kStreamCount - additional_grid_count) {
        grid_size_per_stream = kApproxGridSizePerStream + 1;
        stream_size = grid_size_per_stream * kBlockSize;
      } else {
        grid_size_per_stream = kApproxGridSizePerStream;
        stream_size = kApproxStreamSize;
      }
      stream_ctxs.emplace_back(
          kSize, kBlockSize, kGridSize, kMaxResultsCount, kStreamCount,
          stream_size, grid_size_per_stream);
    }
  }

  void registerPinnedMemory(std::vector<char> &datablocks,
                            std::vector<uint64_t> &seek_indices,
                            accelerator::FilterContext &cond_ctx) {
    cudaCheckError(hipHostRegister(
        &datablocks[0], sizeof(char) * datablocks.size(), hipHostMallocMapped));
    cudaCheckError(hipHostRegister(
        &seek_indices[0], sizeof(uint64_t) * seek_indices.size(),
        hipHostMallocMapped));
    cudaCheckError(hipHostRegister(
        &cond_ctx, sizeof(accelerator::FilterContext), hipHostMallocMapped));
  }

  void unregisterPinnedMemory(std::vector<char> &datablocks,
                              std::vector<uint64_t> &seek_indices,
                              accelerator::FilterContext &cond_ctx) {
    cudaCheckError( hipHostUnregister(&datablocks[0]) );
    cudaCheckError( hipHostUnregister(&seek_indices[0]) );
    cudaCheckError( hipHostUnregister(&cond_ctx) );
  }

  void initParams(const std::vector<char> &datablocks,
                  const std::vector<uint64_t> &seek_indices,
                  const accelerator::FilterContext &cond_ctx) {
    uint64_t start = 0;
    uint64_t start_datablocks = seek_indices[start];
    for (size_t i = 0; i < kStreamCount; ++i) {
      RudaBlockStreamContext &ctx = stream_ctxs[i];

      uint64_t size, size_datablocks;
      if (i == kStreamCount - 1) {
        size = kSize - start;
        size_datablocks = datablocks.size() - start_datablocks;
      } else {
        size = ctx.kStreamSize;
        size_datablocks = seek_indices[start + size] - start_datablocks;
      }

      // Copies sources to GPU (datablocks, seek_indices)
      // Accelerated by stream-pipelining...
      ctx.initParams(
          datablocks, seek_indices, start, size, start_datablocks,
          size_datablocks);

      start += ctx.kStreamSize;
      if (start >= kSize) break;
      start_datablocks = seek_indices[start];
    }
  }

  void populateToCuda(const std::vector<char> &datablocks,
                      const std::vector<uint64_t> &seek_indices,
                      const accelerator::FilterContext &cond_ctx) {
    // Allocation Part
    // Cuda Parameters
    cudaCheckError(hipMalloc(
        (void **) &d_datablocks, sizeof(char) * datablocks.size()));
    total_gpu_used_memory += sizeof(char) * datablocks.size();
    cudaCheckError(hipMalloc(
        (void **) &d_seek_indices, sizeof(uint64_t) * kSize));
    total_gpu_used_memory += sizeof(uint64_t) * kSize;
    cudaCheckError(hipMalloc(
        (void **) &d_cond_ctx, sizeof(accelerator::FilterContext)));
    total_gpu_used_memory += sizeof(accelerator::FilterContext);
    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.cudaMallocGpuBlockSeekStarts();
      total_gpu_used_memory += ctx.total_gpu_used_memory;
    }

    cudaCheckError(hipMemcpy(
        d_cond_ctx, &cond_ctx, sizeof(accelerator::FilterContext),
        hipMemcpyHostToDevice));

    // Asynchronous memory copying
    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.initializeStream();
    }

    // Copies sources to GPU (datablocks, seek_indices)
    // Accelerated by stream-pipelining...
    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.populateToCuda_d_results_idx();
    }

    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.populateToCuda_d_datablocks(datablocks, d_datablocks);
    }

    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.populateToCuda_d_seek_indices(seek_indices, d_seek_indices);
    }

    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.populateToCuda_d_gpu_block_seek_starts();
    }
  }

  void executeKernels(size_t kTotalDataSize) {
    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.executeKernel(
          // Parameters
          kTotalDataSize,
          // Sources
          d_datablocks, d_seek_indices, d_cond_ctx);
    }
  }

  void copyFromCuda() {
    for (RudaBlockStreamContext &ctx : stream_ctxs) {
      ctx.copyFromCuda();
    }
  }

  void translatePairsToSlices(std::vector<rocksdb::Slice> &keys,
                              std::vector<rocksdb::Slice> &values) {
    for (auto &ctx : stream_ctxs) {
      cudaCheckError( hipStreamSynchronize(ctx.stream) );
      for (size_t i = 0; i < ctx.h_results_count; ++i) {
        RudaKVPair &result = ctx.h_results[i];
        size_t key_size = result.key()->size_;
        size_t value_size = result.value()->size_;
        char *key = new char[key_size];
        char *value = new char[value_size];
        memcpy(
            key, result.key()->stack_data_, sizeof(char) * key_size);
        memcpy(
            value, result.value()->stack_data_, sizeof(char) * value_size);
        keys.emplace_back(key, key_size);
        values.emplace_back(value, value_size);
      }
    }
  }

  void log() {
    std::cout << "[CUDA][BlockStreamManager]" << std::endl
        << "kSize: " << kSize << std::endl
        << "kGridSize: " << kGridSize << std::endl
        << "kBlockSize: " << kBlockSize << std::endl
        << "kStreamCount: " << kStreamCount << std::endl
        << "kApproxGridSizePerStream: "
            << kApproxGridSizePerStream << std::endl
        << "kApproxStreamSize: "
            << kApproxStreamSize << std::endl
        << "Max Results Count: " << kMaxResultsCount << std::endl
        << "======================" << std::endl;

    std::cout << "BlockStreamContexts" << std::endl;
    for (size_t i = 0; i < kStreamCount; ++i) {
      RudaBlockStreamContext &ctx = stream_ctxs[i];
      std::cout << "Stream: " << i << std::endl
          << "kStreamSize: " << ctx.kStreamSize << std::endl
          << "kGridSizePerStream: " << ctx.kGridSizePerStream << std::endl
          << "Start Offset: " << ctx.seek_start_offset << std::endl
          << "Size: " << ctx.seek_size << std::endl
          << "Start DataBlocks: " << ctx.datablocks_start_offset << std::endl
          << "Size DataBlocks: " << ctx.datablocks_size << std::endl
          << "_____" << std::endl
          << "Max cache size: " << ctx.kMaxCacheSize << std::endl;
      for (size_t j = 0; j < ctx.kGridSizePerStream; ++j) {
        std::cout << "GPU Block Seek Start[" << j << "]: "
            << ctx.gpu_block_seek_starts[j] << std::endl;
      }
      std::cout << "-----------" << std::endl;
    }
  }

  void freeCudaObjects() {
    cudaCheckError( hipFree(d_datablocks) );
    cudaCheckError( hipFree(d_seek_indices) );
    cudaCheckError( hipFree(d_cond_ctx) );
  }

  void clear() {
    freeCudaObjects();
    for (auto &ctx : stream_ctxs) {
      ctx.clear();
    }
    stream_ctxs.clear();
  }
};

__global__
void kernel::rudaStreamIntBlockFilterKernel(// Parameters (ReadOnly)
                                            size_t offset, size_t kSize,
                                            size_t dataSize,
                                            size_t maxCacheSize,
                                            char *data, uint64_t *seek_indices,
                                            accelerator::FilterContext *ctx,
                                            uint64_t *block_seek_start_indices,
                                            // Variables
                                            unsigned long long int *results_idx,
                                            // Results
                                            RudaKVPair *results) {
  uint64_t i = offset + blockDim.x * blockIdx.x + threadIdx.x;

  // Overflow kernel ptr case.
  if (i >= kSize) {
    return;
  }

  // Shared variables.
  // Caches data used from threads in single block.
  extern __shared__ char cached_data[];

  uint64_t block_seek_start_index = block_seek_start_indices[blockIdx.x];
  uint64_t start = seek_indices[i] - block_seek_start_index;
  uint64_t end = 0;
  if (i == (kSize - 1)) {
    // Last seek index case. 'end' must be end of data.
    end = dataSize - block_seek_start_index;
  } else {
    // 'end' must be next seek index.
    end = seek_indices[i + 1] - block_seek_start_index;
  }

  for (size_t j = start; j < end; ++j) {
    size_t data_idx = block_seek_start_index + j;
    if (data_idx >= dataSize || j >= maxCacheSize) {
      break;
    }
    cached_data[j] = data[data_idx];
  }

  __syncthreads();

  size_t size = end - start;
  DecodeSubDataBlocks(
      // Parameters
      cached_data, size, start, end, ctx,
      // Results
      results_idx, results);
}

int sstStreamIntBlockFilter(std::vector<char> &datablocks,
                            std::vector<uint64_t> &seek_indices,
                            accelerator::FilterContext context,
                            const size_t max_results_count,
                            std::vector<rocksdb::Slice> &keys,
                            std::vector<rocksdb::Slice> &values) {
  RudaBlockStreamManager block_stream_mgr(
      seek_indices.size() /* kSize */,
      64 /* kBlockSize */,
      4 /* kStreamCount */,
      max_results_count);

  // Copy & Initializes variables from host to device.
  block_stream_mgr.initParams(datablocks, seek_indices, context);
  // block_stream_mgr.log();

  block_stream_mgr.registerPinnedMemory(datablocks, seek_indices, context);
  // ----------------------------------------------
  // Cuda Stream Pipelined (Accelerate)
  block_stream_mgr.populateToCuda(datablocks, seek_indices, context);
  std::cout << "MB: " << MB << std::endl;
  std::cout << "Total GPU used memory: "
      << (block_stream_mgr.total_gpu_used_memory / (MB)) << "MB" << std::endl;
  block_stream_mgr.executeKernels(datablocks.size());
  block_stream_mgr.copyFromCuda();
  // ----------------------------------------------
  block_stream_mgr.translatePairsToSlices(keys, values);
  block_stream_mgr.unregisterPinnedMemory(datablocks, seek_indices, context);
  block_stream_mgr.clear();

  return accelerator::ACC_OK;
}

}  // namespace ruda
