#include "hip/hip_runtime.h"

#include <chrono>
#include <cstdio>
#include <functional>
#include <iostream>
#include <string>
#include <thread>
#include <vector>
extern "C" {
#include <pinpool.h>
#include <filemap.h>
}
#include "accelerator/cuda/block_decoder.h"
#include "accelerator/cuda/filter.h"
#include "rocksdb/slice.h"
#include "table/format.h"
#include "stdio.h"

#define KB 1024
#define MB 1024 * KB
#define GB 1024 * MB

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

namespace ruda {
namespace kernel {
__global__
void rudaDonardFilterKernel(char **file_address, uint64_t size, uint64_t *block_index, uint64_t *g_block_index, uint64_t block_unit, uint64_t *handles,
 RudaSchema *schema, unsigned long long int *d_results_idx, donardSlice *d_results);

__global__
void rudaCopyKernel(unsigned long long int count, donardSlice *d_results, char* d_total_results, unsigned long long int *total_results_idx);

__global__
void testKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *total_results_idx);
}  // namespace kernel

struct DonardManager {

  // fileMap list
  struct filemap ** fmlist; 

  // Parameter

  unsigned long long int *num_entries_;

  int num_file_;
  int total_blocks_;
  int block_unit_;
  int num_thread_;
  int max_results_count_;   
  std::vector<uint64_t> gpu_blocks_;

  unsigned long long int results_size;
  unsigned long long int count;
 
  
  // MetaData
  char **file_address;
  uint64_t *block_index; // the number of blocks in each file
  uint64_t *g_block_index;
  uint64_t *d_handles;

  // Schema
  RudaSchema *d_schema; // device schema
  RudaSchema h_schema; // host schema

  // Result
  unsigned long long int *d_results_idx;
  donardSlice *d_results;
  donardSlice * h_results;

  unsigned long long int *total_results_idx;
  char * d_total_results;
 //char * h_total_results;

  DonardManager(int num_file, int total_blocks, int block_unit, int num_thread, int max_results_count) {
    std::cout << "[DONARD MANAGER INITALIZE]" << std::endl;
    num_file_ = num_file;
    total_blocks_ = total_blocks;
    block_unit_ = block_unit;
    num_thread_ = num_thread;
    max_results_count_ = max_results_count;
    results_size = 0;
    count = 0;
  }

  void populate(std::vector<std::string> files, std::vector<uint64_t> num_blocks, std::vector<uint64_t> handles, const rocksdb::SlicewithSchema &schema) {
    std::cout << "[DONARD POPULATE]" << std::endl;
    fmlist = (filemap **)malloc(sizeof(filemap *) * num_file_);
    for(uint i = 0; i < num_file_; i++) {
      fmlist[i] = filemap_open_cuda(files[i].c_str());
    }

    std::cout << "[DONARD POPULATE1] " << num_file_ << " " << total_blocks_ << std::endl;
    //cudaCheckError(hipMalloc((void ***) &file_address, sizeof(char *) * num_file_));
    cudaCheckError(hipHostAlloc((void**)&file_address, sizeof(char*) * num_file_, hipHostMallocMapped));

    std::cout << "[DONARD POPULATE 1-1]" << std::endl;
    for(uint i = 0; i < num_file_; i++) {
      file_address[i] = (char *)fmlist[i]->data;
    }
    
    int tmp_gpu_blocks = 0;
    for(uint i = 0; i < num_blocks.size(); i++) {
      int unit = 0;
      int remain = 0;
      if( i == 0 ) {
        unit = num_blocks[i] / block_unit_;
        remain = num_blocks[i] % block_unit_;
      } else {
        unit = (num_blocks[i] - num_blocks[i-1]) / block_unit_;
        remain = (num_blocks[i] - num_blocks[i-1]) % block_unit_;
      }
      tmp_gpu_blocks += unit;
      if(remain != 0) tmp_gpu_blocks += 1;
      gpu_blocks_.emplace_back(tmp_gpu_blocks);
    }

    std::cout << "gpu block index size " << gpu_blocks_.size() << std::endl;

    std::cout << "[DONARD POPULATE2] block_index " << sizeof(uint64_t) * num_blocks.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &block_index, sizeof(uint64_t) * num_blocks.size()));
    cudaCheckError(hipMemcpy(block_index, &num_blocks[0], sizeof(uint64_t) * num_blocks.size(), hipMemcpyHostToDevice));

    std::cout << "[DONARD POPULATE2] g_block_index " << sizeof(uint64_t) * gpu_blocks_.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &g_block_index, sizeof(uint64_t) * gpu_blocks_.size()));
    cudaCheckError(hipMemcpy(g_block_index, &gpu_blocks_[0], sizeof(uint64_t) * gpu_blocks_.size(), hipMemcpyHostToDevice));

    std::cout << "[DONARD POPULATE2] d_handles " << sizeof(uint64_t) * handles.size() << std::endl;
    cudaCheckError(hipMalloc((void **) &d_handles, sizeof(uint64_t) * handles.size()));
    cudaCheckError(hipMemcpy(d_handles, &handles[0], sizeof(uint64_t) * handles.size(), hipMemcpyHostToDevice));
  
    std::cout << "[DONARD POPULATE3]" << std::endl;
    // Deep copy for Schema
    rocksdb::SlicewithSchema* copy_schema = schema.clone();
    cudaCheckError(hipHostRegister(&h_schema, sizeof(rocksdb::SlicewithSchema), hipHostMallocMapped));
    cudaCheckError(hipMalloc((void **) &d_schema, sizeof(RudaSchema)));
    cudaCheckError(h_schema.populateToCuda(*copy_schema));
    cudaCheckError(hipMemcpy(d_schema, &h_schema, sizeof(RudaSchema), hipMemcpyHostToDevice));

    std::cout << "[DONARD POPULATE4]" << std::endl;
    cudaCheckError(hipMalloc((void **) &d_results_idx, sizeof(unsigned long long int)));
    cudaCheckError(hipMemset(d_results_idx, 0, sizeof(unsigned long long int)));

    std::cout << "[DONARD POPULATE5] d_results " << sizeof(donardSlice) * max_results_count_ << std::endl;
    cudaCheckError(hipMalloc((void **) &d_results, sizeof(donardSlice) * max_results_count_));

    cudaCheckError(hipMalloc((void **) &total_results_idx, sizeof(unsigned long long int)));
    cudaCheckError(hipMemset(total_results_idx, 0, sizeof(unsigned long long int)));
  
  }

  void executeKernel() {
    std::cout << "[DONARD KERNEL EXECUTE] : " << gpu_blocks_.back() << std::endl;
    kernel::rudaDonardFilterKernel<<< gpu_blocks_.back(), num_thread_ >>> (file_address, num_file_, block_index, g_block_index, block_unit_, d_handles,
                             d_schema, d_results_idx, d_results);
 
    hipDeviceSynchronize();
    num_entries_ = (unsigned long long int *)malloc(sizeof(unsigned long long int));
    cudaCheckError(hipMemcpy(num_entries_, d_results_idx, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
    
    count = *num_entries_;
    h_results = (donardSlice *)malloc(sizeof(donardSlice) * count);
    cudaCheckError(hipMemcpy(h_results, d_results, sizeof(donardSlice) * count, hipMemcpyDeviceToHost));

    for(uint i = 0; i <count ; i++) {
      results_size += h_results[i].key_size + h_results[i].d_size + 4;
    }

    std::cout << " results_size : " << results_size << std::endl;
    cudaCheckError(hipMalloc((void **) &d_total_results, sizeof(char) * results_size));

    uint32_t blockGrid = count / num_thread_ ;
    uint32_t remain = count % num_thread_ ;   
    if (remain != 0) blockGrid += 1;

    std::cout << " blockGrid : " << blockGrid << " count : " << count << std::endl;
    kernel::rudaCopyKernel<<< blockGrid , num_thread_ >>> (count, d_results, d_total_results, total_results_idx);
    hipDeviceSynchronize();
  }

  void translatePairsToSlices(std::vector<rocksdb::PinnableSlice> &keys, std::vector<rocksdb::PinnableSlice> &results, char **data_buf, uint64_t *num_entries) {

    std::cout << "[DONARD TRANSLATE TO SLICES 0]" << std::endl;
    //h_total_results = (char *)malloc(sizeof(char) * results_size);
    //cudaCheckError(hipMemcpy(h_total_results, d_total_results, sizeof(char) * results_size, hipMemcpyDeviceToHost));

    *num_entries = count;
    *data_buf = (char *)malloc(sizeof(char) * results_size);
    char *target_ptr = *data_buf;
    cudaCheckError(hipMemcpy(target_ptr, d_total_results, sizeof(char) * results_size, hipMemcpyDeviceToHost));
  
    /*
    std::cout << "[DONARD TRANSLATE TO SLICES 1]" << std::endl;  
    std::cout << "[DONARD TRANSLATE TO SLICES 2] " << count << std::endl;

    char *initialPtr = h_total_results;
    for (size_t i = 0; i < count; i++) {
      size_t key_size = *((unsigned short *)initialPtr);
      initialPtr += 2;
      size_t value_size = *((unsigned short *)initialPtr);
      initialPtr += 2;

      keys.emplace_back(std::move(rocksdb::PinnableSlice(initialPtr, key_size)));
      initialPtr += key_size;

      results.emplace_back(std::move(rocksdb::PinnableSlice(initialPtr, value_size)));
      initialPtr += value_size;
    }
   */
  }

  void clear() {
    std::cout << "[DONARD CLEAR]" << std::endl;
    for(uint i = 0; i < num_file_; i++) {
      filemap_free(fmlist[i]);
    } 
    cudaCheckError(hipHostFree(file_address)); 
    cudaCheckError(hipFree(block_index));
    cudaCheckError(hipFree(g_block_index));
    cudaCheckError(hipFree(d_handles));

    cudaCheckError(h_schema.clear());
    cudaCheckError(hipFree(d_schema));

    cudaCheckError(hipFree(d_results_idx));
    cudaCheckError(hipFree(d_results));
    cudaCheckError(hipFree(total_results_idx));
    cudaCheckError(hipFree(d_total_results));

    free(h_results);
   // free(h_total_results);
  }
};

__global__
void kernel::rudaDonardFilterKernel(char **file_address, uint64_t size, uint64_t *block_index, uint64_t *g_block_index, uint64_t g_block_unit, uint64_t * d_handles,
 RudaSchema *schema, unsigned long long int *results_idx, donardSlice *d_results) {  
  
  // blockDim.x * blockIdx.x + threadIdx.x;
  // blockDim = number of Thread in block

  // Find file location 
  unsigned int idx = getFileIdx(blockIdx.x, size, g_block_index);

  int gBlockOffset = (idx == 0) ? blockIdx.x : blockIdx.x - g_block_index[idx-1];
  int accumulatedBlocks = (idx == 0) ? g_block_unit * gBlockOffset : block_index[idx-1] + g_block_unit * gBlockOffset;

  int gBlockRemain = 0;
  if (blockIdx.x == g_block_index[idx] - 1) {
    gBlockRemain = (idx == 0) ? block_index[idx] % g_block_unit : (block_index[idx] - block_index[idx -1]) % g_block_unit;
  }
  if (gBlockRemain != 0) g_block_unit = gBlockRemain; 

  char *filePtr = file_address[idx];  
  const char *startPtr = (gBlockOffset == 0) ? filePtr : filePtr + d_handles[accumulatedBlocks -1];

  uint32_t blockSize = 0;
  int kDataBlockIndexTypeBitShift = 31;
  uint32_t kNumRestartsMask = (1u << kDataBlockIndexTypeBitShift) - 1u;
  uint32_t kBlockTrailerSize = 5;

  uint32_t threadsPerBlock = blockDim.x / g_block_unit;
  uint32_t threadRemain = blockDim.x % g_block_unit;
  uint32_t threadIdInBlock = threadIdx.x / g_block_unit;
  uint32_t blockLocation = threadIdx.x % g_block_unit; 

  if (blockLocation < threadRemain) threadsPerBlock += 1;

  if (gBlockOffset == 0) {
   if(blockLocation == 0) {
    blockSize = d_handles[accumulatedBlocks] - kBlockTrailerSize;
   } else {
    blockSize = d_handles[accumulatedBlocks + blockLocation] - d_handles[accumulatedBlocks + blockLocation - 1] - kBlockTrailerSize;
    startPtr += d_handles[accumulatedBlocks + blockLocation - 1];
   }
  } else { 
    blockSize = d_handles[accumulatedBlocks + blockLocation] - d_handles[accumulatedBlocks + blockLocation - 1] - kBlockTrailerSize; 
    startPtr += d_handles[accumulatedBlocks + blockLocation - 1] - d_handles[accumulatedBlocks - 1];
  }

  uint32_t numRestarts = DecodeFixed32(startPtr + blockSize - sizeof(uint32_t));
  numRestarts = numRestarts & kNumRestartsMask;
  uint32_t restartOffset = static_cast<uint32_t>(blockSize) - (1 + numRestarts) * sizeof(uint32_t);

  if (numRestarts < threadIdInBlock + 1) return;

  uint32_t numTask = numRestarts / threadsPerBlock;
  uint32_t remainNumTask = numRestarts % threadsPerBlock;

  bool lastThread = false;
  if (numTask == 0 && threadIdInBlock == numRestarts - 1) lastThread = true;
  if (numTask != 0 && threadIdInBlock == threadsPerBlock - 1) lastThread = true; 

  if (threadIdInBlock < remainNumTask) numTask += 1;

  if(numTask == 0) return;

  uint32_t startLocation = restartOffset;

  startLocation += (threadIdInBlock >= remainNumTask) ? (remainNumTask + (numTask * threadIdInBlock)) * sizeof(uint32_t) : (numTask * threadIdInBlock * sizeof(uint32_t));

  if (!lastThread) restartOffset = 0;
  DecodeNFilterOnSchemaDonard(startPtr, restartOffset, startLocation, numTask, schema, results_idx, d_results); 

}

__global__
void kernel::rudaCopyKernel(unsigned long long int count, donardSlice *d_results, char* total_results, unsigned long long int *total_results_idx) {

  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx >= count) {
    return;
  }
  //printf("blockidx : %d, threadidx : %d\n", blockIdx.x, threadIdx.x);
  size_t key_size = d_results[idx].key_size;
  size_t value_size = d_results[idx].d_size;
  unsigned long long int kvPairSize = key_size + value_size;

  unsigned long long int resultOffset = atomicAdd(total_results_idx, kvPairSize + 4);

  char* targetIdx = total_results + resultOffset;

  char *k_size = (char *)&key_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = k_size[i];
  }

  targetIdx += 2;

  char *v_size = (char *)&value_size;
  for (uint i = 0; i < sizeof(unsigned short); i++) {
    targetIdx[i] = v_size[i];
  }

  targetIdx += 2;

  for(uint i = 0; i < key_size; i++) {
    targetIdx[i] = d_results[idx].key[i];
  }

  targetIdx += key_size;

  for(uint i = 0; i < value_size; i++) {
    targetIdx[i] = d_results[idx].d_data[i];
  }  
}

__global__
void kernel::testKernel(unsigned long long int count, donardSlice *d_results, unsigned long long int *total_results_idx) {
  unsigned long long int idx = blockDim.x * blockIdx.x + threadIdx.x;
  printf("idx : %d\n", idx);
}

int donardFilter( std::vector<std::string> files, std::vector<uint64_t> num_blocks, std::vector<uint64_t> handles, const rocksdb::SlicewithSchema &schema,
                  uint64_t max_results_count,
                  std::vector<rocksdb::PinnableSlice> &keys,
                  std::vector<rocksdb::PinnableSlice> &results, char **data_buf, uint64_t *num_entries) {

  std::cout << "[GPU][donardFilter] START" << std::endl;

  void *warming_up;
  cudaCheckError(hipMalloc(&warming_up, 0));
  cudaCheckError(hipFree(warming_up));

  DonardManager donard_mgr(
      files.size(),
      num_blocks.back(),
      30,
      128 /* kBlockSize */,
      max_results_count);

  donard_mgr.populate(files, num_blocks, handles, schema);
  donard_mgr.executeKernel();

  donard_mgr.translatePairsToSlices(keys, results, data_buf, num_entries);
  donard_mgr.clear();

  std::cout << "This is end " << std::endl;
  hipDeviceSynchronize();
  hipDeviceReset();
  return accelerator::ACC_OK;
}

}  // namespace ruda
