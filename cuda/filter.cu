#include "hip/hip_runtime.h"


#include <algorithm>
#include <cstdio>
#include <iostream>
#include <limits>
#include <string>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include "cuda/block_decoder.h"
#include "cuda/filter.h"
#include "rocksdb/slice.h"
#include "table/format.h"

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

namespace ruda {

namespace kernel {
  // Kernels
  __global__
  void rudaIntBlockFilterKernel(// Parameters (ReadOnly)
                                size_t kSize, size_t dataSize,
                                size_t resultsCount, char *data,
                                uint64_t *seek_indices, ConditionContext *ctx,
                                uint64_t *block_seek_start_indices,
                                // Variables
                                unsigned long long int *results_idx,
                                // Results
                                ruda::RudaSlice * results_keys,
                                ruda::RudaSlice * results_values);
  __global__
  void rudaPopulateSlicesFromHeap(size_t kSize, RudaSlice *sources);
  __global__
  void rudaIntFilterKernel(ConditionContext *context, int *values,
                          int *results);
}  // namespace kernel

struct RudaIntTransformator {
  ConditionContext _context;

  RudaIntTransformator(ConditionContext context) {
    this->_context = context;
  }

  __host__ __device__
  int operator()(const int target) const {
    switch (this->_context._op) {
      case EQ:
        return target == this->_context._pivot ? 1 : 0;
      case LESS:
        return target < this->_context._pivot ? 1 : 0;
      case GREATER:
        return target > this->_context._pivot ? 1 : 0;
      case LESS_EQ:
        return target <= this->_context._pivot ? 1 : 0;
      case GREATER_EQ:
        return target >= this->_context._pivot ? 1 : 0;
      default:
        return 0;
    }
  }
};

struct RudaBlockFilterContext {
  // Parameters
  char *d_datablocks;
  uint64_t *d_seek_indices;
  ConditionContext *d_cond_context;
  uint64_t *d_block_seek_start_indices;

  // Results - Device
  unsigned long long int *d_results_idx;   // Atomic incrementer index
  RudaSlice *d_results_keys;   // Filtered keys
  RudaSlice *d_results_values; // Filtered values

  // Results - Host
  // Total results count copied from 'd_results_idx' after kernel call.
  unsigned long long int h_results_count;
  RudaSlice *h_results_keys;   // Filtered keys
  RudaSlice *h_results_values; // Filtered values

  // Cuda Kernel Parameters
  const size_t kSize = 0;
  const int kBlockSize = 0;
  const int kGridSize = 0;
  const size_t kMaxResultsCount = 0;
  size_t kMaxCacheSize = 0;

  RudaBlockFilterContext(const size_t total_size, const int block_size,
                         const size_t max_results_count)
      : kSize(total_size), kBlockSize(block_size),
        kGridSize(ceil((float) total_size / (float) block_size)),
        kMaxResultsCount(max_results_count) {}

  size_t CalculateBlockSeekIndices(const std::vector<uint64_t> &seek_indices) {
    uint64_t block_seek_start_indices[kGridSize];

    for (size_t i = 0; i < kGridSize; ++i) {
      size_t i_thread = i * kBlockSize;
      block_seek_start_indices[i] = seek_indices[i_thread];
    }

    size_t max_cache_size = 0;
    for (size_t i = 0; i < kGridSize; ++i) {
      if (i == kGridSize - 1) {
        break;
      }
      size_t cache_size =
          block_seek_start_indices[i + 1] - block_seek_start_indices[i];
      if (cache_size > max_cache_size) {
        max_cache_size = cache_size;
      }
    }

    cudaCheckError(hipMemcpy(
        d_block_seek_start_indices, block_seek_start_indices,
        sizeof(uint64_t) * kGridSize, hipMemcpyHostToDevice));
    return sizeof(char) * max_cache_size;
  }

  void populateParametersToCuda(const std::vector<char> &datablocks,
                                const std::vector<uint64_t> &seek_indices,
                                const ConditionContext &cond_context) {
    // Cuda Parameters
    cudaCheckError(hipMalloc(
        (void **) &d_datablocks, sizeof(char) * datablocks.size()));
    cudaCheckError(hipMalloc(
        (void **) &d_seek_indices, sizeof(uint64_t) * kSize));
    cudaCheckError(hipMalloc(
        (void **) &d_cond_context, sizeof(ConditionContext)));
    cudaCheckError(hipMalloc(
        (void **) &d_block_seek_start_indices, sizeof(uint64_t) * kGridSize));
    kMaxCacheSize = CalculateBlockSeekIndices(seek_indices);

    // Cuda Results
    cudaCheckError(hipMalloc(
        (void **) &d_results_idx, sizeof(unsigned long long int)));
    cudaCheckError(hipMalloc(
        (void **) &d_results_keys, sizeof(RudaSlice) * kMaxResultsCount));
    cudaCheckError(hipMalloc(
        (void **) &d_results_values, sizeof(RudaSlice) * kMaxResultsCount));

    cudaCheckError(hipMemcpy(
        d_datablocks, &datablocks[0], sizeof(char) * datablocks.size(),
        hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(
        d_seek_indices, &seek_indices[0], sizeof(uint64_t) * kSize,
        hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(
        d_cond_context, &cond_context, sizeof(ConditionContext),
        hipMemcpyHostToDevice));
    cudaCheckError(hipMemset(
        d_results_idx, 0, sizeof(unsigned long long int)));
  }

  void populateResultsFromCuda() {
    // Copy to host results
    cudaCheckError(hipMemcpy(
        &h_results_count, d_results_idx, sizeof(unsigned long long int),
        hipMemcpyDeviceToHost));
    h_results_keys = new RudaSlice[h_results_count];
    h_results_values = new RudaSlice[h_results_count];
    cudaCheckError(hipMemcpy(
        h_results_keys, d_results_keys, sizeof(RudaSlice) * h_results_count,
        hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(
        h_results_values, d_results_values, sizeof(RudaSlice) * h_results_count,
        hipMemcpyDeviceToHost));

    // Populates results from cuda heap space.
    for (size_t i = 0; i < h_results_count; ++i) {
      cudaCheckError(hipMalloc(
          (void **) &h_results_keys[i].data_,
          sizeof(char) * h_results_keys[i].size()));
      cudaCheckError(hipMalloc(
          (void **) &h_results_values[i].data_,
          sizeof(char) * h_results_values[i].size()));
    }

    cudaCheckError(hipFree(d_results_keys));
    cudaCheckError(hipFree(d_results_values));
    cudaCheckError(hipMalloc(
        (void **) &d_results_keys, sizeof(RudaSlice) * h_results_count));
    cudaCheckError(hipMalloc(
        (void **) &d_results_values, sizeof(RudaSlice) * h_results_count));

    cudaCheckError(hipMemcpy(
        d_results_keys, h_results_keys, sizeof(RudaSlice) * h_results_count,
        hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(
        d_results_values, h_results_values, sizeof(RudaSlice) * h_results_count,
        hipMemcpyHostToDevice));

    size_t kResultsGridSize = ceil(
        (float) h_results_count / (float) kBlockSize);
    kernel::rudaPopulateSlicesFromHeap<<<kResultsGridSize, kBlockSize>>> (
        h_results_count, d_results_keys);
    kernel::rudaPopulateSlicesFromHeap<<<kResultsGridSize, kBlockSize>>> (
        h_results_count, d_results_values);

    cudaCheckError(hipMemcpy(
        h_results_keys, d_results_keys, sizeof(RudaSlice) * h_results_count,
        hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(
        h_results_values, d_results_values, sizeof(RudaSlice) * h_results_count,
        hipMemcpyDeviceToHost));
  }

  void copyToFinalResults(std::vector<rocksdb::Slice> &keys,
                          std::vector<rocksdb::Slice> &values) {
    // Copy to results
    for (size_t i = 0; i < h_results_count; ++i) {
      size_t key_size = h_results_keys[i].size();
      size_t value_size = h_results_values[i].size();
      char *key = new char[key_size];
      char *value = new char[value_size];
      cudaCheckError(hipMemcpy(
          key, h_results_keys[i].data(), key_size, hipMemcpyDeviceToHost));
      cudaCheckError(hipMemcpy(
          value, h_results_values[i].data(), value_size,
          hipMemcpyDeviceToHost));
      keys.emplace_back(rocksdb::Slice(key, h_results_keys[i].size()));
      values.emplace_back(rocksdb::Slice(value, h_results_keys[i].size()));
    }
  }

  void freeParametersFromCuda() {
    cudaCheckError(hipFree(d_datablocks));
    cudaCheckError(hipFree(d_seek_indices));
    cudaCheckError(hipFree(d_cond_context));
    cudaCheckError(hipFree(d_block_seek_start_indices));
  }

  void freeResultsFromCuda() {
    cudaCheckError(hipFree(d_results_idx));
    cudaCheckError(hipFree(d_results_keys));
    cudaCheckError(hipFree(d_results_values));

    // Free 2d cuda array
    for (size_t i = 0; i < h_results_count; ++i) {
      if (h_results_keys[i].size() != 0) {
        cudaCheckError(hipFree(h_results_keys[i].data()));
      }
      if (h_results_values[i].size() != 0) {
        cudaCheckError(hipFree(h_results_values[i].data()));
      }
    }

    delete[] h_results_keys;
    delete[] h_results_values;
  }

  void freeAllFromCuda() {
    freeParametersFromCuda();
    freeResultsFromCuda();
  }
};

__global__
void kernel::rudaIntBlockFilterKernel(// Parameters (ReadOnly)
                                      size_t kSize, size_t dataSize,
                                      size_t resultsCount, char *data,
                                      uint64_t *seek_indices,
                                      ConditionContext *ctx,
                                      uint64_t *block_seek_start_indices,
                                      // Variables
                                      unsigned long long int *results_idx,
                                      // Results
                                      ruda::RudaSlice * results_keys,
                                      ruda::RudaSlice * results_values) {
  uint64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  // Overflow kernel ptr case.
  if (i >= kSize) {
    return;
  }

  // Shared variables.
  // Caches data used from threads in single block.
  extern __shared__ char cached_data[];

  uint64_t block_seek_start_index = block_seek_start_indices[blockIdx.x];
  uint64_t start = seek_indices[i] - block_seek_start_index;
  uint64_t end = 0;
  if (i == (kSize - 1)) {
    // Last seek index case. 'end' must be end of data.
    end = dataSize - block_seek_start_index;
  } else {
    // 'end' must be next seek index.
    end = seek_indices[i + 1] - block_seek_start_index;
  }

  for (size_t j = start; j < end; ++j) {
    size_t data_idx = block_seek_start_index + j;
    cached_data[j] = data[data_idx];
  }

  __syncthreads();

  size_t size = end - start;
  DecodeSubDataBlocks(
      // Parameters
      cached_data, size, start, end, ctx,
      // Results
      results_idx, results_keys, results_values);
}

__global__
void kernel::rudaPopulateSlicesFromHeap(size_t kSize, RudaSlice *sources) {
  uint64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= kSize || sources[i].data() == nullptr) {
    return;
  }

  sources[i].populateDataFromHeap();
  delete sources[i].heap_data_;
}

__global__
void kernel::rudaIntFilterKernel(ConditionContext *context, int *values,
                         int *results) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  switch (context->_op) {
    case EQ:
      results[index] = values[index] == context->_pivot ? 1 : 0;
      break;
    case LESS:
      results[index] = values[index] < context->_pivot ? 1 : 0;
      break;
    case GREATER:
      results[index] = values[index] > context->_pivot ? 1 : 0;
      break;
    case LESS_EQ:
      results[index] = values[index] <= context->_pivot ? 1 : 0;
      break;
    case GREATER_EQ:
      results[index] = values[index] >= context->_pivot ? 1 : 0;
      break;
    default:
      break;
  }
}

int sstIntFilter(const std::vector<int> &values,
                 const ConditionContext context,
                 std::vector<int> &results) {
  rocksdb::BlockContents block;
  // std::cout << "[RUDA][sstIntFilter] Start" << std::endl;
  results.resize(values.size());

  // std::cout << "[sstIntFilter] Inputs" << std::endl;
  // std::cout << "[sstIntFilter] Inputs - values" << std::endl;
  // for (int i = 0; i < values.size(); ++i) {
    // std::cout << values[i] << " ";
  // }
  // std::cout << std::endl;
  // std::cout << "[sstIntFilter] Inputs - context: " << context.toString()
      // << std::endl;

  thrust::device_vector<int> d_values(values);
  thrust::device_vector<int> d_results(values.size());

  RudaIntTransformator rudaTrans(context);
  thrust::transform(d_values.begin(), d_values.end(), d_results.begin(),
                    rudaTrans);

  // std::cout << "[sstIntFilter] Results" << std::endl;
  // std::cout << "[sstIntFilter] Results - d_results" << std::endl;
  // for (int i = 0; i < d_results.size(); ++i) {
    // std::cout << d_results[i] << " ";
  // }
  // std::cout << std::endl;

  thrust::copy(d_results.begin(), d_results.end(), results.begin());
  // std::cout << "[sstIntFilter] Results - results" << std::endl;
  // for (int i = 0; i < results.size(); ++i) {
    // std::cout << results[i] << " ";
  // }
  // std::cout << std::endl;

  return ruda::RUDA_OK;
}

int sstIntNativeFilter(const std::vector<int> &values,
                       const ConditionContext context,
                       std::vector<int> &results) {
  int *d_values, *d_results;
  int *h_results;
  ConditionContext *d_context;
  const int kSize = values.size();
  const int kBlockSize = 256;
  const float kGridSize = ceil((float) kSize / (float) kBlockSize);

  h_results = (int *) malloc(sizeof(int) * kSize);

  hipMalloc((void **) &d_values, sizeof(int) * kSize);
  hipMalloc((void **) &d_context, sizeof(ConditionContext));
  hipMalloc((void **) &d_results, sizeof(int) * kSize);

  hipMemcpy(d_values, &values[0], sizeof(int) * kSize, hipMemcpyHostToDevice);
  hipMemcpy(d_context, &context, sizeof(ConditionContext),
             hipMemcpyHostToDevice);

  kernel::rudaIntFilterKernel<<<kGridSize, kBlockSize>>>(
      d_context, d_values, d_results);

  hipMemcpy(h_results, d_results, sizeof(int) * kSize, hipMemcpyDeviceToHost);

  hipFree(d_values);
  hipFree(d_context);
  hipFree(d_results);

  results.assign(h_results, h_results + kSize);
  free(h_results);

  return ruda::RUDA_OK;
}

int sstIntBlockFilter(const std::vector<char> &datablocks,
                      const std::vector<uint64_t> &seek_indices,
                      const ConditionContext context,
                      const size_t max_results_count,
                      std::vector<rocksdb::Slice> &keys,
                      std::vector<rocksdb::Slice> &values) {
  RudaBlockFilterContext block_context(
      seek_indices.size() /* kSize */,
      64 /* kBlockSize */,
      max_results_count);

  // Copy & Initializes variables from host to device.
  block_context.populateParametersToCuda(datablocks, seek_indices, context);

  std::cout
      << "[BlockContext]" << std::endl
      << "kSize: " << block_context.kSize << std::endl
      << "kGridSize: " << block_context.kGridSize << std::endl
      << "kBlockSize: " << block_context.kBlockSize << std::endl
      << "kMaxCacheSize: " << block_context.kMaxCacheSize << std::endl
      << "DataSize: " << datablocks.size() << std::endl
      << "Max Results Count: " << block_context.kMaxResultsCount << std::endl;

  cudaCheckError(hipDeviceSetLimit(
      hipLimitMallocHeapSize, 100 * sizeof(char) * datablocks.size()));

  // Call kernel.
  kernel::rudaIntBlockFilterKernel<<<block_context.kGridSize,
                                     block_context.kBlockSize,
                                     block_context.kMaxCacheSize>>>(
      // Kernel Parameters
      block_context.kSize, datablocks.size(),
      block_context.kMaxResultsCount, block_context.d_datablocks,
      block_context.d_seek_indices, block_context.d_cond_context,
      block_context.d_block_seek_start_indices,
      // Kernel Variables
      block_context.d_results_idx,
      // Kernel Results
      block_context.d_results_keys, block_context.d_results_values);

  block_context.populateResultsFromCuda();
  block_context.copyToFinalResults(keys, values);

  std::cout << "[BlockContext::Result]" << std::endl
      << "Total Results Count: " << block_context.h_results_count << std::endl;

  // Free device variables.
  block_context.freeAllFromCuda();

  return ruda::RUDA_OK;
}

}  // namespace ruda
