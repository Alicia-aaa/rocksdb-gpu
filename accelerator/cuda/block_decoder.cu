#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "accelerator/cuda/block_decoder.h"
#include "accelerator/cuda/cuda_util.h"

#define NUM_TABLE_BYTES 4
#define DEFAULT_KEY_BUF_SIZE 16

namespace ruda {

// Note(totoro): This implementation copied from 'block.h', 'coding.h',
//               'coding.cc' to use on cuda codes.
// Helper routine: decode the next block entry starting at "p",
// storing the number of shared key bytes, non_shared key bytes,
// and the length of the value in "*shared", "*non_shared", and
// "*value_length", respectively.  Will not derefence past "limit".
//
// If any errors are detected, returns nullptr.  Otherwise, returns a
// pointer to the key delta (just past the three decoded values).
struct DecodeEntry {

  __device__
  const char* GetVarint32PtrFallback(const char* p, const char* limit,
                                     uint32_t* value) {
    uint32_t result = 0;
    for (uint32_t shift = 0; shift <= 28 && p < limit; shift += 7) {
      uint32_t byte = *(reinterpret_cast<const unsigned char*>(p));
      p++;
      if (byte & 128) {
        // More bytes are present
        result |= ((byte & 127) << shift);
      } else {
        result |= (byte << shift);
        *value = result;
        return reinterpret_cast<const char*>(p);
      }
    }
    return nullptr;
  }

  __device__
  const char* GetVarint32Ptr(const char* p,
                             const char* limit,
                             uint32_t* value) {
    if (p < limit) {
      uint32_t result = *(reinterpret_cast<const unsigned char*>(p));
      if ((result & 128) == 0) {
        *value = result;
        return p + 1;
      }
    }
    return GetVarint32PtrFallback(p, limit, value);
  }

  __device__
  const char* operator()(const char* p, const char* limit,
                         uint32_t* shared, uint32_t* non_shared,
                         uint32_t* value_length) {
    // We need 2 bytes for shared and non_shared size. We also need one more
    // byte either for value size or the actual value in case of value delta
    // encoding.
    // assert(limit - p >= 3);
    *shared = reinterpret_cast<const unsigned char*>(p)[0];
    *non_shared = reinterpret_cast<const unsigned char*>(p)[1];
    *value_length = reinterpret_cast<const unsigned char*>(p)[2];
    if ((*shared | *non_shared | *value_length) < 128) {
      // Fast path: all three values are encoded in one byte each
      p += 3;
    } else {
      if ((p = GetVarint32Ptr(p, limit, shared)) == nullptr) return nullptr;
      if ((p = GetVarint32Ptr(p, limit, non_shared)) == nullptr) return nullptr;
      if ((p = GetVarint32Ptr(p, limit, value_length)) == nullptr) {
        return nullptr;
      }
    }

    // Using an assert in place of "return null" since we should not pay the
    // cost of checking for corruption on every single key decoding
    // assert(!(static_cast<uint32_t>(limit - p) < (*non_shared + *value_length)));
    return p;
  }
};

__host__ __device__
uint32_t DecodeFixed32(const char* ptr) {
  // if (port::kLittleEndian) {
  //   // Load the raw bytes
  //   uint32_t result;
  //   memcpy(&result, ptr, sizeof(result));  // gcc optimizes this to a plain load
  //   return result;
  // }
  return ((static_cast<uint32_t>(static_cast<unsigned char>(ptr[0])))
      | (static_cast<uint32_t>(static_cast<unsigned char>(ptr[1])) << 8)
      | (static_cast<uint32_t>(static_cast<unsigned char>(ptr[2])) << 16)
      | (static_cast<uint32_t>(static_cast<unsigned char>(ptr[3])) << 24));
}

__host__ __device__
uint64_t DecodeFixed64(const char* ptr) {
  // if (port::kLittleEndian) {
  //   // Load the raw bytes
  //   uint64_t result;
  //   memcpy(&result, ptr, sizeof(result));  // gcc optimizes this to a plain load
  //   return result;
  // }
  uint64_t lo = DecodeFixed32(ptr);
  uint64_t hi = DecodeFixed32(ptr + 4);
  return (hi << 32) | lo;
}

__device__
unsigned long long int atomicAggInc(unsigned long long int *counter) {
  auto g = cooperative_groups::coalesced_threads();
  unsigned long long int warp_res;
  if (g.thread_rank() == 0) {
    warp_res = atomicAdd(counter, g.size());
  }
  return g.shfl(warp_res, 0) + g.thread_rank();
}

__device__
void DecodeNFilterSubDataBlocks(// Parameters
                                const char *cached_data,
                                const uint64_t cached_data_size,
                                const uint64_t block_offset,
                                const uint64_t start_idx,
                                const uint64_t end_idx,
                                accelerator::FilterContext *ctx,
                                // Results
                                unsigned long long int *results_idx,
                                ruda::RudaKVIndexPair *results) {
  const char *subblock = &cached_data[start_idx];
  const char *limit = &cached_data[end_idx];
  while (subblock < limit) {
    uint32_t shared, non_shared, value_size;
    subblock = DecodeEntry()(subblock, limit, &shared, &non_shared,
                             &value_size);
    const char *key;
    size_t key_size;
    if (shared == 0) {
      key = subblock;
      key_size = non_shared;
    } else {
      // TODO(totoro): We need to consider 'shared' data within subblock.
      key = subblock;
      key_size = shared + non_shared;
    }

    const char *value = subblock + non_shared;
    uint64_t decoded_value = DecodeFixed64(value);
    bool filter_result = false;
    switch (ctx->_op) {
      case accelerator::EQ:
        filter_result = decoded_value == ctx->_pivot;
        break;
      case accelerator::LESS:
        filter_result = decoded_value < ctx->_pivot;
        break;
      case accelerator::GREATER:
        filter_result = decoded_value > ctx->_pivot;
        break;
      case accelerator::LESS_EQ:
        filter_result = decoded_value <= ctx->_pivot;
        break;
      case accelerator::GREATER_EQ:
        filter_result = decoded_value >= ctx->_pivot;
        break;
      default:
        break;
    }
    if (filter_result) {
      unsigned long long int idx = atomicAdd(results_idx, 1);
      size_t key_start = key - cached_data;
      size_t value_start = value - cached_data;
      results[idx] = RudaKVIndexPair(
          block_offset + key_start,
          block_offset + key_start + key_size,
          block_offset + value_start,
          block_offset + value_start + value_size);
    }

    // Heap Version
    // char *results_key = new char[key_size];
    // char *results_value = new char[value_size];
    // memcpy(results_key, key, key_size);
    // memcpy(results_value, value, value_size);
    // results_keys[idx] = RudaSlice(results_key, key_size);
    // results_values[idx] = RudaSlice(results_value, value_size);

    // Next DataKey...
    subblock = value + value_size;
  }
}

__device__
void CachedDecodeNFilterOnSchema(// Parameters
                                 const char *cached_data,
                                 const uint64_t cached_data_size,
                                 const uint64_t block_offset,
                                 const uint64_t start_idx,
                                 const uint64_t end_idx,
                                 RudaSchema *schema,
                                 // Results
                                 unsigned long long int *results_idx,
                                 ruda::RudaKVIndexPair *results) {
  const char *subblock = cached_data + start_idx;
  const char *limit = cached_data + end_idx;
  size_t key_buf_size = DEFAULT_KEY_BUF_SIZE;
  size_t key_buf_length = 0;
  char *key_buf = new char[key_buf_size];
  while (subblock < limit) {
    uint32_t shared, non_shared, value_size;
    subblock = DecodeEntry()(subblock, limit, &shared, &non_shared,
                             &value_size);
    const char *key;
    size_t key_size;
    if (shared == 0) {
      key = subblock;
      key_size = non_shared;
      if (key_size > key_buf_size) {
        delete[] key_buf;
        key_buf_size = key_size;
        key_buf = new char[key_buf_size];
      }
      memset(key_buf, 0, sizeof(char) * key_buf_size);
      memcpy(key_buf, key, sizeof(char) * key_size);
      key_buf_length = key_size;
    } else {
      key = subblock;
      key_size = shared + non_shared;
      if (key_size > key_buf_size) {
        char *new_key_buf = new char[key_size];
        memcpy(new_key_buf, key_buf, sizeof(char) * shared);
        delete[] key_buf;
        key_buf_size = key_size;
        key_buf = new_key_buf;
      }
      memcpy(key_buf + shared, key, sizeof(char) * non_shared);
      key_buf_length = key_size;
    }

    const char *value = subblock + non_shared;

    bool is_equal_to_schema = true;
    for (size_t i = 0; i < NUM_TABLE_BYTES; ++i) {
      if (key_buf[i] != schema->data[i]) {
        is_equal_to_schema = false;
        break;
      }
    }
    if (!is_equal_to_schema) {
      subblock = value + value_size;
      continue;
    }

    long decoded_value = rudaConvertRecord(schema, value);
    bool filter_result = false;
    switch (schema->ctx._op) {
      case accelerator::EQ:
        filter_result = decoded_value == schema->ctx._pivot;
        break;
      case accelerator::LESS:
        filter_result = decoded_value < schema->ctx._pivot;
        break;
      case accelerator::GREATER:
        filter_result = decoded_value > schema->ctx._pivot;
        break;
      case accelerator::LESS_EQ:
        filter_result = decoded_value <= schema->ctx._pivot;
        break;
      case accelerator::GREATER_EQ:
        filter_result = decoded_value >= schema->ctx._pivot;
        break;
      case accelerator::INVALID:
        // INVALID case, return all data to result.
        filter_result = true;
        break;
      default:
        break;
    }
    if (filter_result) {
      unsigned long long int idx = atomicAdd(results_idx, 1);
      size_t value_start = value - cached_data;
      results[idx] = RudaKVIndexPair(
          block_offset + value_start,
          block_offset + value_start + value_size);
    }

    // Next DataKey...
    subblock = value + value_size;
  }

  delete[] key_buf;
}

__device__
void DecodeNFilterOnSchema(// Parameters
                           const char *data,
                           const uint64_t lookup_size,
                           const uint64_t block_offset,
                           const uint64_t start_idx,
                           const uint64_t end_idx,
                           RudaSchema *schema,
                           // Results
                           unsigned long long int *results_idx,
                           ruda::RudaKVIndexPair *results) {
  const char *subblock = data + block_offset + start_idx;
  const char *limit = data + block_offset + end_idx;
  size_t key_buf_size = DEFAULT_KEY_BUF_SIZE;
  size_t key_buf_length = 0;
  char *key_buf = new char[key_buf_size];
  while (subblock < limit) {
    uint32_t shared, non_shared, value_size;
    subblock = DecodeEntry()(subblock, limit, &shared, &non_shared,
                             &value_size);
    const char *key;
    size_t key_size;
    if (shared == 0) {
      key = subblock;
      key_size = non_shared;
      if (key_size > key_buf_size) {
        delete[] key_buf;
        key_buf_size = key_size;
        key_buf = new char[key_buf_size];
      }
      memset(key_buf, 0, sizeof(char) * key_buf_size);
      memcpy(key_buf, key, sizeof(char) * key_size);
      key_buf_length = key_size;
    } else {
      key = subblock;
      key_size = shared + non_shared;
      if (key_size > key_buf_size) {
        char *new_key_buf = new char[key_size];
        memcpy(new_key_buf, key_buf, sizeof(char) * shared);
        delete[] key_buf;
        key_buf_size = key_size;
        key_buf = new_key_buf;
      }
      memcpy(key_buf + shared, key, sizeof(char) * non_shared);
      key_buf_length = key_size;
    }

    const char *value = subblock + non_shared;

    bool is_equal_to_schema = true;
    for (size_t i = 0; i < NUM_TABLE_BYTES; ++i) {
      if (key_buf[i] != schema->data[i]) {
        is_equal_to_schema = false;
        break;
      }
    }
    if (!is_equal_to_schema) {
      subblock = value + value_size;
      continue;
    }

    long decoded_value = rudaConvertRecord(schema, value);
    bool filter_result = false;
    switch (schema->ctx._op) {
      case accelerator::EQ:
        filter_result = decoded_value == schema->ctx._pivot;
        break;
      case accelerator::LESS:
        filter_result = decoded_value < schema->ctx._pivot;
        break;
      case accelerator::GREATER:
        filter_result = decoded_value > schema->ctx._pivot;
        break;
      case accelerator::LESS_EQ:
        filter_result = decoded_value <= schema->ctx._pivot;
        break;
      case accelerator::GREATER_EQ:
        filter_result = decoded_value >= schema->ctx._pivot;
        break;
      case accelerator::INVALID:
        // INVALID case, return all data to result.
        filter_result = true;
        break;
      default:
        break;
    }
    if (filter_result) {
      unsigned long long int idx = atomicAdd(results_idx, 1);
      size_t value_start = value - data;
      results[idx] = RudaKVIndexPair(value_start, value_start + value_size);
    }

    // Next DataKey...
    subblock = value + value_size;
  }

  delete[] key_buf;
}

}  // namespace ruda
