#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>


using namespace std;

int main() {
  size_t count = 0;
  size_t size = 64 * 1024 * 1024 * sizeof(float);
  while (true) {
    void *ptr;
    while (posix_memalign(&ptr, 4096, size) != 0) {
      cout << "posix_memalign failed at " << count * 256 << " MB" << endl;
    }
    cout << "ptr = " << ptr << endl;
    while (hipHostRegister(ptr, size, 0) != hipSuccess) {
      cout << "hipHostRegister failed at " << count * 256 << " MB" << endl;
    }
    count++;
    cout << "Allocated " << count * 256 << " MB" << endl;
  }
}
